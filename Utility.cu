#include <random>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "Utility.cuh"


namespace Utility {
	const char* cublasGetErrorString(hipblasStatus_t status)
	{
	    switch(status)
	    {
		case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
		case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
		case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
		case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
		case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
		case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
		case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
		case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
	    }
	    return "Unknown cuBLAS error";
	}

	Gradients::Gradients(double* deviceVisiblePositiveGradients_, double* deviceHiddenPositiveGradients_, double* deviceVisibleNegativeGradients_, double* deviceHiddenNegativeGradients_) :
		deviceVisiblePositiveGradients(deviceVisiblePositiveGradients_), 
		deviceHiddenPositiveGradients(deviceHiddenPositiveGradients_), 
		deviceVisibleNegativeGradients(deviceVisibleNegativeGradients_), 
		deviceHiddenNegativeGradients(deviceHiddenNegativeGradients_) {}

	Gradients::~Gradients() {
		hipFree(deviceHiddenPositiveGradients);
		hipFree(deviceVisibleNegativeGradients);
		hipFree(deviceHiddenNegativeGradients);
	}

	Deltas::Deltas(double* deviceDeltaWeights_, double* deviceDeltaBiasesVisible_, double* deviceDeltaBiasesHidden_) :
		deviceDeltaWeights(deviceDeltaWeights_), 
		deviceDeltaBiasesVisible(deviceDeltaBiasesVisible_), 
		deviceDeltaBiasesHidden(deviceDeltaBiasesHidden_) {}

	Deltas::~Deltas() {
		hipFree(deviceDeltaWeights);
		hipFree(deviceDeltaBiasesVisible);
		hipFree(deviceDeltaBiasesHidden);
	}
}
