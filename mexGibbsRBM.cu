#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <string>
#include <thrust/device_ptr.h>

#include "mex.h"
#include "gpu/mxGPUArray.h"

#include "RestrictedBoltzmannMachine.cuh"
#include "Utility.cuh"


void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{   
    
    std::string modeType(mxArrayToString(prhs[4]));
    std::string modeWork(mxArrayToString(prhs[5]));
    double* params;
    params = mxGetPr(prhs[6]); int hidden = params[0];
    params = mxGetPr(prhs[7]); int steps = params[0];
    
    double* data = mxGetPr(prhs[0]);
    int visible = mxGetN(prhs[0]);
    int samples = mxGetM(prhs[0]);
    
    double* weights = mxGetPr(prhs[1]);
    double* biasesVisible = mxGetPr(prhs[2]);
    double* biasesHidden = mxGetPr(prhs[3]);

    RestrictedBoltzmannMachine* rbm;
    if (modeType == "LinearNRelu") 
    	rbm = new LinearNReluRestrictedBoltzmannMachine(weights, biasesVisible, biasesHidden, hidden, visible);    
    else if (modeType == "BinaryBinary")    
	rbm = new BinaryBinaryRestrictedBoltzmannMachine(weights, biasesVisible, biasesHidden, hidden, visible);
    else if (modeType == "LinearBinary")    
	rbm = new LinearBinaryRestrictedBoltzmannMachine(weights, biasesVisible, biasesHidden, hidden, visible);
    else 
	mexErrMsgTxt("Incorrect mode parameters. You can pass the one from the following set {BinaryBinary, LinearBinary, LinearNRelu}"); 

    double* result;
    int n; 
    if (modeWork == "Sample") {
    	 result = rbm->GibbsChainIteration(data, samples, steps, "Sample");
 	 n = visible; 
    }
    else if (modeWork == "Reconstruction") {    
	 result = rbm->GibbsChainIteration(data, samples, 1, "Reconstruction");
	 n = hidden;
    }
    else 
	mexErrMsgTxt("Incorrect mode parameters. You can pass the one from the following set {Sample, Reconstruction}"); 
  
    plhs[0] = mxCreateDoubleMatrix(samples, n, mxREAL);
    double* resultOutput = mxGetPr(plhs[0]);

    for (int i = 0; i < samples*n; i++)
        resultOutput[i] = result[i];

    delete rbm;
    hipDeviceReset();
}
